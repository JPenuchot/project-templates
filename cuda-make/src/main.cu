
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstddef>
#include <vector>

template<typename T>
__global__ void iota(T* in, T* out)
{
  out[threadIdx.x] = in[threadIdx.x] * threadIdx.x;
}

template <typename T>
struct cuda_managed_alloc
{
  using value_type = T;
  using size_type = std::size_t;
  using difference_type = std::ptrdiff_t;

  static T* allocate(std::size_t n)
  {
    T* res;
    hipMallocManaged((void**)&res, n * sizeof(T));
    return res;
  }

  static void deallocate(T* ptr, std::size_t)
  {
    hipFree(ptr);
  }
};

template<typename T>
using cuda_vector = std::vector<T, cuda_managed_alloc<T>>;

int main(int, char const *[])
{
  std::vector<float, cuda_managed_alloc<float>> v(1024);

  for(auto& elmt : v)
    elmt = 1;

  iota<<<1, v.size()>>>(v.data(), v.data());

  hipDeviceSynchronize();

  for(auto& elmt : v)
    std::cout << elmt << '\n';

  return 0;
}
